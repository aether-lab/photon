#include "hip/hip_runtime.h"
/*
  For more information, please see: http://software.sci.utah.edu

  The MIT License

  Copyright (c) 2012-2013
  Scientific Computing and Imaging Institute, University of Utah

  License for the specific language governing rights and limitations under
  Permission is hereby granted, free of charge, to any person obtaining a
  copy of this software and associated documentation files (the "Software"),
  to deal in the Software without restriction, including without limitation
  the rights to use, copy, modify, merge, publish, distribute, sublicense,
  and/or sell copies of the Software, and to permit persons to whom the
  Software is furnished to do so, subject to the following conditions:

  The above copyright notice and this permission notice shall be included
  in all copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
  OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
  THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
  FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
  DEALINGS IN THE SOFTWARE.
*/

#define DEBUG 0

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <teem/nrrd.h>


// includes, GL
#include "opengl_include.h"
#include <float.h>
#include <assert.h>

// includes
#include "cutil.h"
#include "cutil_math.h"
#include <hip/hip_runtime.h>
#include "cuda_gl_interop.h"
#include <cstdlib>

#include "RenderParameters.h"
#include "kernel_render.h"
#include "kernel_filter.h"

RenderParameters* dparams;
hipArray* data_array = 0, *texture_array = 0, *color_array = 0;
unsigned int last_width, last_height;
unsigned int* d_out = 0;
float4* d_inout = 0;
float* d_rand_x = 0;

void Host_CopyMemory(RenderParameters* params);
void Host_Resize(RenderParameters* paramsp);

__host__ int rgbToIntHost(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

__host__ float getRandom()
{
  return drand48();
}

extern "C"
{

    void Host_Init(RenderParameters* paramsp)
    {
        hipMalloc((void**)&dparams, sizeof(RenderParameters));
        hipMemcpy(dparams, paramsp, sizeof(RenderParameters),hipMemcpyHostToDevice);

        //setup data texture
        tex_data.addressMode[0] = hipAddressModeClamp;
        tex_data.addressMode[1] = hipAddressModeClamp;
        tex_data.filterMode = hipFilterModeLinear;
        tex_data.normalized = false;

        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
        hipExtent extent = make_hipExtent(paramsp->data_width, paramsp->data_height, paramsp->data_depth);
        checkCudaErrors( hipMalloc3DArray(&data_array, &channelDesc, extent) );
        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr = make_hipPitchedPtr((void*)paramsp->data, extent.width*sizeof(float4), extent.width, extent.height);
        copyParams.dstArray = data_array;
        copyParams.extent = extent;
        copyParams.kind = hipMemcpyHostToDevice;
        checkCudaErrors(  hipMemcpy3D(&copyParams) );

        hipBindTextureToArray(tex_data, data_array, channelDesc);

        //setup cutoff texture
        if (paramsp->cutoff  == CUTOFF_IMAGE)
        {
            tex_cutoff.addressMode[0] = hipAddressModeClamp;
            tex_cutoff.addressMode[1] = hipAddressModeClamp;
            tex_cutoff.filterMode = hipFilterModeLinear;
            tex_cutoff.normalized = true;

            hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
            hipMallocArray(&texture_array, &channelDesc2, paramsp->cutoffSize.x, paramsp->cutoffSize.y);
            hipMemcpyToArray(texture_array, 0, 0, paramsp->cutoff_rgb, paramsp->cutoffSize.x*paramsp->cutoffSize.y*sizeof(float4), hipMemcpyHostToDevice);

            hipBindTextureToArray(tex_cutoff, texture_array, channelDesc2);
            paramsp->cutoff_dirty = false;
        }

        Host_Resize(paramsp);
    }

    void Host_Render(RenderParameters* paramsp)
    {
        printf("rendering...");
        if (last_width != paramsp->width || last_height != paramsp->height)
            Host_Resize(paramsp);

        RenderParameters& params = *paramsp;
        Host_CopyMemory(paramsp);

        printf("min_bound : %f, %f, %f\n", params.min_bound.x, params.min_bound.y, params.min_bound.z);
        printf("max_bound : %f, %f, %f\n", params.max_bound.x, params.max_bound.y, params.max_bound.z);

        // define block and grid by the CUDA kernel
        dim3 block(16,16,1);
        dim3 grid((params.width/block.x),(params.height/block.y),1);   //positions go 0 to 100, which maps to -1 to 1 on each lightplace axis

        // compute four random numbers to calculate initial position of photon
        paramsp->rand1 = drand48();
        paramsp->rand2 = drand48();
        paramsp->rand3 = drand48();
        paramsp->rand4 = drand48();
        
        // allocate space on the device for the results
        hipMemcpy(dparams, paramsp, sizeof(RenderParameters),hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        
        for(int i =0; i < paramsp->numRenderPasses; i++)
        {
            printf("i : %d",i);
            // render image
            kernel_render<<< grid, block>>>(dparams, d_inout, d_out, d_rand_x);
            hipDeviceSynchronize();
            paramsp->passes+=paramsp->raysPerPixel;
        }

        // register photons and smooth image
        kernel_filter<<< grid, block>>>(dparams, d_inout, d_out);
        
        // copy image array to CPU
        hipMemcpy(paramsp->out_rgb, d_out, sizeof(unsigned int)*params.width*params.height, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        // display image on screen
        //glDrawPixels(params.width, params.height, GL_RGBA, GL_UNSIGNED_BYTE, paramsp->out_rgb);
        
        printf(" rendering finished\n");
    }

    void Host_Clear(RenderParameters* paramsp)
    {
        if (!d_inout)
            return;
        hipMemcpy(d_inout, paramsp->inout_rgb, sizeof(float4)*paramsp->width*paramsp->height, hipMemcpyHostToDevice);
    }

    void Host_Kill()
    {
        /* This functions sets all pointers to zero and frees GPU memory*/
        
        RenderParameters* dparams;
        hipArray* data_array = 0, *texture_array = 0, *color_array = 0;
        hipFree(d_inout);
        hipFree(dparams);
        hipFree(d_out);
        hipFree(d_rand_x);

        hipUnbindTexture (tex_data);
        checkCudaErrors (hipFreeArray (data_array));
        hipUnbindTexture (tex_data2);
        checkCudaErrors (hipFreeArray (texture_array));
        hipUnbindTexture (tex_cutoff);
        checkCudaErrors (hipFreeArray (color_array));
    }


}

void Host_CopyMemory(RenderParameters* paramsp)
{
    //TODO: NOTE: for debugging perposes only memcopy, however need to support size changes
    if (paramsp->cutoff_dirty)
    {
        hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
        hipMemcpyToArray(texture_array, 0, 0, paramsp->cutoff_rgb, paramsp->cutoffSize.x*paramsp->cutoffSize.y*sizeof(float4), hipMemcpyHostToDevice);
        paramsp->cutoff_dirty = false;
    }
}

void Host_Resize(RenderParameters* paramsp)
{
    printf("resizing to %d %d \n", paramsp->width, paramsp->height);
    paramsp->passes = 0;
    int window_size = paramsp->width*paramsp->height;
    if (d_inout)
        hipFree(d_inout);

    hipMalloc((void**)&d_inout, sizeof(float4)*paramsp->width*paramsp->height);
    for(size_t i = 0; i < paramsp->width*paramsp->height; i++)
        paramsp->inout_rgb[i] = make_float4(0,0,0,0);
    hipMemcpy(d_inout, paramsp->inout_rgb, sizeof(float4)*paramsp->width*paramsp->height, hipMemcpyHostToDevice);

    if (d_out)
        hipFree(d_out);
    hipMalloc((void**)&d_out, sizeof(unsigned int)*paramsp->width*paramsp->height);
    last_width = paramsp->width;
    last_height = paramsp->height;

    if (paramsp->out_rgb)
        free(paramsp->out_rgb);
    paramsp->out_rgb = (unsigned int*)malloc(sizeof(unsigned int)*paramsp->width*paramsp->height);

    if (d_rand_x)
        hipFree(d_rand_x);

    paramsp->random_array  = (float*)malloc(sizeof(float)*window_size);

    for(int i =0;i<window_size;i++)
        paramsp->random_array[i] = getRandom();

    hipMalloc((void**)&d_rand_x, window_size*sizeof(float));
    hipMemcpy(d_rand_x, paramsp->random_array,sizeof(float)*window_size, hipMemcpyHostToDevice);
}

